#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_fp16.h"

#include <torch/extension.h>

__global__ void my_attention_kernel(
    half* _o,
    half* _q,
    half* _k,
    half* _v,
    half* _k_cache,
    half* _v_cache,
    int _batch,
    int _tokens
)
{

}

torch::Tensor my_attention(
    torch::Tensor q,
    torch::Tensor k,
    torch::Tensor v,
    torch::Tensor k_cache,
    torch::Tensor v_cache
)
{
    auto batch = q.size(0);
    auto hidden_dim = q.size(2);
    auto tokens = k_cache.size(1);
    auto options = torch::TensorOptions().dtype(torch::kFloat16).device(torch::kCUDA, 0);
    torch::Tensor o = torch::full({batch, hidden_dim}, 0, options);

    half* q_ptr = reinterpret_cast<half*>(q.data_ptr<at::Half>());
    half* k_ptr = reinterpret_cast<half*>(k.data_ptr<at::Half>());
    half* v_ptr = reinterpret_cast<half*>(v.data_ptr<at::Half>());
    half* k_cache_ptr = reinterpret_cast<half*>(k_cache.data_ptr<at::Half>());
    half* v_cache_ptr = reinterpret_cast<half*>(v_cache.data_ptr<at::Half>());

    half* o_ptr = reinterpret_cast<half*>(o.data_ptr<at::Half>());

    // Invoke your kernel:
    // my_attention_kernel <<<?, ?>>> (o_ptr, q_ptr, k_ptr, v_ptr, k_cache_ptr, v_cache_ptr, batch, tokens, ...);

    return o;

}